/**
    fftconvolve2d.cu
    Compute real-complex FFT convolutions on the GPU

    @author: Qiang Wang (qiangwang@comp.hkbu.edu.hk)
    @version: 1.0 05/12/2018
*/


#include "cuda_utils.h"
#include "fft_utils.cu"
#include "fftconvolve2d.cuh"
#include <chrono>
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime_api.h>
#include <numeric>


#define DEBUG 0
#define VERBOSE 0
#define QDFFT_MEASURE 1
#define NUM_ITERATIONS 100

using namespace std;

//extern "C" {
    std::unordered_map<std::string, std::vector<double>> timerMap;

    // index array for qdfft
    int* input_xs = NULL;
    int* input_ys = NULL;
    int* output_xs = NULL;
    int* output_ys = NULL;
    int* d_input_xs = NULL;
    int* d_input_ys = NULL;
    int* d_output_xs = NULL;
    int* d_output_ys = NULL;
    int* p = NULL;
    int* s = NULL;

    int* sparse_input_xs = NULL;
    int* sparse_input_ys = NULL;
    int* begin_arr = NULL;
    int* end_arr = NULL;
    int* qdV_arr = NULL;
    int non_zero = 0;
    int non_empty_qdV = 0;

    int* d_sparse_input_xs = NULL;
    int* d_sparse_input_ys = NULL;
    int* d_begin_arr = NULL;
    int* d_end_arr = NULL;
    int* d_qdV_arr = NULL;
    hipfftComplex *d_qdVectors = NULL;
    hipfftComplex* d_w_results = NULL;
    hipfftHandle plan1d; 

    void reset_timer()
    {
        timerMap.clear();
    }

    void report_timer()
    {
        double total_time = 0;
        if (timerMap.find("total_time") != timerMap.end())
            total_time = std::accumulate(timerMap["total_time"].begin() + 1,
                                         timerMap["total_time"].end(), 0.0);
        for (auto &kv : timerMap) {
            auto v = kv.second;
            double sum = std::accumulate(v.begin() + 1, v.end(), 0.0);
            printf("[%s] Calls: %d, Average time: %.3lf ms, Global Percentage: %.2lf %%\n",
                   kv.first.c_str(), (int)v.size(), sum / (int)v.size(), 100.0 * sum / total_time);
        }
    }


    struct complexMultiplier
    {
        double scale;
        complexMultiplier(double scale): scale(scale) {};

        __host__ __device__
        hipfftComplex operator() (const hipfftComplex &v1,
                                       const hipfftComplex &v2) const
        {
            hipfftComplex res;
            res.x = (v1.x * v2.x - v1.y * v2.y) * scale;
            res.y = (v1.x * v2.y + v1.y * v2.x) * scale;
            return res;
        }
    };

    struct complexAdder
    {
        double scale;
        complexAdder(double scale): scale(scale) {};

        __host__ __device__
        hipfftComplex operator() (const hipfftComplex &v1,
                                       const hipfftComplex &v2) const
        {
            hipfftComplex res;
            res.x = (v1.x + v2.x) * scale;
            res.y = (v1.y + v2.y) * scale;
            return res;
        }
    };

    /**
        Creates an FFT Plan if it has not been yet initialized

        @plan: Pointer to the plan that will be created/initialized
        @size: Size of the FFT for which this plan will be used
        @type: Type of the FFT
        @batch: Number of FFTs of the specified size that will be computed together.

    */
    void create_plan(hipfftHandle *plan, size_t nRows, size_t nCols, hipfftType type)
    {
        size_t workSize;
        int ret = hipfftGetSize(*plan, &workSize);
        if (ret == HIPFFT_INVALID_PLAN) {
            if (hipfftPlan2d(plan, nRows, nCols, type) != HIPFFT_SUCCESS) {
                fprintf(stderr, "CUFFT error: Plan creation failed");
            }
        }
    }

    // helper function for QD fft
    void gcd(int p, int s, int& x, int& y, int& r){
	if(p == 0){
	    x = 0;
	    y = 1;
	    r = s;
	    return;
	}
	if(s == 0){
	    x = 1;
	    y = 0;
	    r = p;
	    return;
	}
	gcd(s, p % s, x, y, r);
	int temp = x;
	x = y;
	y = temp - p / s * y;
	return;
    }

    void find_input_index(int p, int s, int t, int N, int* idx_x, int* idx_y){
	int a0 = 0, b0 = 0, gcd_value = 0;
	gcd(p, s, a0, b0, gcd_value);
	a0 = a0 * t;
	b0 = b0 * t;
	//printf("find gcd: p:%d, s:%d, a:%d, b:%d, gcd:%d.\n", p, s, a0, b0, gcd_value);
	for(int i = 0;i<N;i++){
	    idx_x[i] = (a0 + s / gcd_value * i) % N;
	    idx_y[i] = (b0 - p / gcd_value * i + N * N) % N;
	}
    }


    /**
        Computes the FFT convolution of two padded signals, direct fft

        @signal: The first signal. This is a pointer to host(CPU) memory
        @signalSize: The signal size
        @kernel: The second signal, also called kernel. This is a pointer to
                 host(CPU) memory
        @kernelSize: The kernel size
        @result: Pointer to host(CPU) memory that contains the convolution result.
                 Sufficient memory ((singalSize + kernelSize -1) * sizeof(hipfftDoubleComplex))
                 has to be allocated before calling the function.
        @d_in: Pointer to GPU memory used by the function. The size of the memory region
                has to be at least 2 * (signalSize + kernelSize - 1)
        @fwplan: An integer handle used to store the forward FFT plan.
        @bwplan: An integer handle used to store the backward FFT plan.
    */
    void convolve_direct_fft(float * Signal, float * Kernels, int kernelCount, 
                             int nRows, int nCols, int kRows, int kCols, float * results)
    {

        hipfftHandle fwplan_signal, fwplan_kernels, bwplan; 
        // timer timer, globalTimer;
        // globalTimer.restart();

        int fftRows = nRows + kRows - 1;
        int fftCols = nCols + kCols - 1;
        int dataSize = nRows * nCols;
        int kernelSize = kRows * kCols;
        int fftSize = fftRows * fftCols;
        int cfftSize = fftRows * (fftCols / 2 + 1);

        // parameters for hipfftPlanMany
        int idist = fftRows * fftCols;
        int odist = fftRows * (fftCols / 2 + 1);
        int inembed[] = {fftRows, fftCols};
        int onembed[] = {fftRows, fftCols / 2 + 1};
        int n[2] = {fftRows, fftCols};

#if DEBUG
        for(int i = 0;i<nRows;i++){
            for(int j = 0;j<nCols;j++)
                printf("%f ", Signal[i * nCols + j]);
            printf("\n");
        }
        printf("\n");
        for(int i = 0;i<kRows;i++){
            for(int j = 0;j<kCols;j++)
                printf("%f ", Kernels[i * kCols + j]);
            printf("\n");
	}
        printf("\n");
#endif

        // initialize device memory
        float *d_signal = NULL;   // device storage of input signal
        float *d_kernels = NULL;  // device storage of kernels and final results
        float *d_paddedSignal = NULL;
        float *d_paddedKernels = NULL;  // device storage of kernels and final results
        float *d_results = NULL;  
        hipfftComplex *d_signal_fft;
        hipfftComplex *d_kernels_fft;
        
        hipMalloc((void**)&d_signal, sizeof(float) * dataSize);
        hipMalloc((void**)&d_kernels, sizeof(float) * kernelSize * kernelCount);
        hipMalloc((void**)&d_paddedSignal, sizeof(float) * fftSize);
        hipMalloc((void**)&d_paddedKernels, sizeof(float) * fftSize * kernelCount);
        hipMalloc((void**)&d_results, sizeof(float) * fftSize * kernelCount);
        hipMalloc((void**)&d_signal_fft, cfftSize * sizeof(hipfftComplex));
        hipMalloc((void**)&d_kernels_fft, cfftSize * kernelCount * sizeof(hipfftComplex));

        // timer.restart();
        hipMemset(d_signal, 0, dataSize * sizeof(float));
        hipMemset(d_kernels, 0, kernelSize * kernelCount * sizeof(float));
        hipMemset(d_paddedSignal, 0, fftSize * sizeof(float));
        hipMemset(d_paddedKernels, 0, fftSize * kernelCount * sizeof(float));
        hipMemset(d_results, 0, fftSize * kernelCount * sizeof(float));
        //hipMemset(d_signal_fft, 0, odist * sizeof(hipfftComplex));
        //hipMemset(d_kernels_fft, 0, odist * kernelCount * sizeof(hipfftComplex));

        // timerMap["memset"].push_back(timer.elapsed());

        // timer.restart();
        // timerMap["memcpy"].push_back(timer.elapsed());

        // padded signal
        hipMemcpy(d_signal, Signal, dataSize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_kernels, Kernels, kernelSize * kernelCount * sizeof(float), hipMemcpyHostToDevice);
        dim3 threadBlock(16, 16, 4);
        dim3 dataBlockGrid(iDivUp(fftCols, threadBlock.x), iDivUp(fftRows, threadBlock.y), 1);
        dim3 kernelBlockGrid(iDivUp(fftCols, threadBlock.x), iDivUp(fftRows, threadBlock.y), iDivUp(kernelCount, threadBlock.z));
        //dim3 kernelBlockGrid(iDivUp(kCols, threadBlock.x), iDivUp(kRows, threadBlock.y), iDivUp(kernelCount, threadBlock.z));
        padDataWithZeros<<<dataBlockGrid, threadBlock>>>(d_paddedSignal, d_signal, fftCols, fftRows, nCols, nRows, 1); 
        padDataWithZeros<<<kernelBlockGrid, threadBlock>>>(d_paddedKernels, d_kernels, fftCols, fftRows, kCols, kRows, kernelCount);

#if VERBOSE
        CudaCheckError();
        float  *h_paddedSignal = (float*)malloc(sizeof(float) * fftSize);  // padded signal
        float  *h_paddedKernels = (float*)malloc(sizeof(float) * fftSize * kernelCount);  // padded kernels

        hipMemcpy(h_paddedSignal, d_paddedSignal, sizeof(float) * fftSize, hipMemcpyDeviceToHost);
        hipMemcpy(h_paddedKernels, d_paddedKernels, sizeof(float) * fftSize * kernelCount, hipMemcpyDeviceToHost);

        for(int i = 0;i<10;i++){
            for(int j = 0;j<10;j++)
            	printf("%f ", h_paddedSignal[i * fftCols + j]);
            printf("\n");
	}
        printf("\n");
        for(int i = 0;i<10;i++){
            for(int j = 0;j<10;j++)
            	printf("%f ", h_paddedKernels[i * fftCols + j]);
            printf("\n");
	}
        printf("\n");
#endif
        hipfftPlan2d(&fwplan_signal, fftRows, fftCols, HIPFFT_R2C);
        hipfftPlanMany(&fwplan_kernels, 2, n, inembed, 1, idist, onembed, 1, odist, HIPFFT_R2C, kernelCount);
        hipfftPlanMany(&bwplan, 2, n, onembed, 1, odist, inembed, 1, idist, HIPFFT_C2R, kernelCount);
        // timer.restart();
        // timerMap["create_plans"].push_back(timer.elapsed());

	// warm up
	{
            // timer.restart();
            hipfftExecR2C(fwplan_signal, d_paddedSignal, d_signal_fft);
            hipfftExecR2C(fwplan_kernels, d_paddedKernels, d_kernels_fft);
            // timerMap["forward"].push_back(timer.elapsed());

            // timer.restart();

            thrust::device_ptr<hipfftComplex> thr_signal_fft(d_signal_fft);
            thrust::device_ptr<hipfftComplex> thr_kernel_fft(d_kernels_fft);
            //thrust::transform(thr_signal_fft, thr_signal_fft + kernelCount * cfftSize, thr_kernel_fft, thr_kernel_fft, complexMultiplier(1.0 / fftSize));
            for(int i = 0 ;i < kernelCount; i++){
                thrust::transform(thr_signal_fft, thr_signal_fft + cfftSize, thr_kernel_fft + i * cfftSize, thr_kernel_fft + i * cfftSize,
                                  complexMultiplier(1.0 / fftSize));
                // timerMap["multiply"].push_back(timer.elapsed());
            }

#if VERBOSE
            size_t freeMem, totalMem;
            hipMemGetInfo(&freeMem, &totalMem);
            std::cout << "Free: " << freeMem << ", Total: " << totalMem << std::endl;
#endif

            // timer.restart();
            hipfftExecC2R(bwplan, d_kernels_fft, d_results);
            //hipfftExecZ2Z(*bwplan, d_kernels_fft, d_kernels, HIPFFT_BACKWARD);
            // timerMap["backward"].push_back(timer.elapsed());
	}

        hipDeviceSynchronize();
	int fft_time = 0;
	int dot_product_time = 0;
	int ifft_time = 0;

	    int duplicate = 8;
		hipfftComplex* d_double_signal_fft;
        hipMalloc((void**)&d_double_signal_fft, cfftSize * duplicate * sizeof(hipfftComplex));
        int num_repeats = NUM_ITERATIONS;
        for (int i = 0; i < num_repeats; ++i) 
	{
            // timer.restart();
            auto start = std::chrono::steady_clock::now();
            hipfftExecR2C(fwplan_signal, d_paddedSignal, d_signal_fft);
            hipfftExecR2C(fwplan_kernels, d_paddedKernels, d_kernels_fft);
            hipDeviceSynchronize();
            auto end = std::chrono::steady_clock::now();
            fft_time += static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() );
            // timerMap["forward"].push_back(timer.elapsed());

            // timer.restart();

            start = std::chrono::steady_clock::now();

			//// use thrust
			//for(int c = 0; c < duplicate;c++)
            //    hipMemcpy(d_double_signal_fft + c * cfftSize, d_signal_fft, sizeof(hipfftComplex) * cfftSize, hipMemcpyDeviceToDevice);
            ////thrust::device_ptr<hipfftComplex> thr_signal_fft(d_signal_fft);
            //thrust::device_ptr<hipfftComplex> thr_signal_fft(d_double_signal_fft);
            //thrust::device_ptr<hipfftComplex> thr_kernel_fft(d_kernels_fft);
            ////thrust::transform(thr_signal_fft, thr_signal_fft + kernelCount * cfftSize, thr_kernel_fft, thr_kernel_fft, complexMultiplier(1.0 / fftSize));
            //for(int i = 0 ;i < kernelCount / duplicate; i++){
            //    thrust::transform(thr_signal_fft, thr_signal_fft + cfftSize * duplicate, thr_kernel_fft + i * duplicate * cfftSize, thr_kernel_fft + i * duplicate * cfftSize,
            //                      complexMultiplier(1.0 / fftSize));
            //    // timerMap["multiply"].push_back(timer.elapsed());
            //}

            dim3 dotBlock(32, 32);
            dim3 dotGrid(fftCols / 32, iDivUp((fftRows / 2 + 1), 32), 16);
			dotProduct<<<dotGrid, dotBlock>>>(d_signal_fft, fftCols, fftRows / 2 + 1, 1.0 / fftSize, kernelCount/16, d_kernels_fft);

            hipDeviceSynchronize();
            end = std::chrono::steady_clock::now();
            dot_product_time += static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() );

#if VERBOSE
            size_t freeMem, totalMem;
            hipMemGetInfo(&freeMem, &totalMem);
            std::cout << "Free: " << freeMem << ", Total: " << totalMem << std::endl;
#endif

            // timer.restart();
            start = std::chrono::steady_clock::now();
            hipfftExecC2R(bwplan, d_kernels_fft, d_results);
            //hipfftExecZ2Z(*bwplan, d_kernels_fft, d_kernels, HIPFFT_BACKWARD);
            // timerMap["backward"].push_back(timer.elapsed());
            hipDeviceSynchronize();
            end = std::chrono::steady_clock::now();
            ifft_time += static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() );
	}

        int avg_fft_time = fft_time / num_repeats;
        int avg_dot_product_time = dot_product_time / num_repeats;
        int avg_ifft_time = ifft_time / num_repeats;
	int avg_total_time = avg_fft_time + avg_dot_product_time + avg_ifft_time;
	printf("direct_fft total time:%d\n", avg_total_time);

        // timer.restart();
        hipMemcpy(results, d_results, kernelCount * fftSize * sizeof(float), hipMemcpyDeviceToHost);
       
        // timerMap["copy_back"].push_back(timer.elapsed());
        // timerMap["total_time"].push_back(globalTimer.elapsed());

        // free GPU memory
        hipFree(d_signal);
        hipFree(d_paddedSignal);
        hipFree(d_kernels);
        hipFree(d_paddedKernels);
        hipFree(d_results);
        hipFree(d_signal_fft);
        hipFree(d_kernels_fft);

    }

    struct saxpy_functor
    {
        const float a;
    
        saxpy_functor(float _a) : a(_a) {}
    
        __host__ __device__
            float operator()(const float& x, const float& y) const { 
                return a * x + y;
            }
    };
    
    void saxpy_fast(float A, thrust::device_ptr<float>& X, thrust::device_ptr<float>& Y, int matrixSize)
    {
        // Y <- A * X + Y
        thrust::transform(X, X + matrixSize, Y, Y, saxpy_functor(A));
    }



    void convolve_turbo_fft_v1(float * Signal, float * Kernels, int kernelCount, 
                             int nRows, int nCols, int kRows, int kCols, float * results)
    {

        hipfftHandle fwplan_signal, fwplan_kernels, bwplan; 
        // timer timer, globalTimer;
        // globalTimer.restart();

        int fftRows = nRows + kRows - 1;
        int fftCols = nCols + kCols - 1;

        int dataSize = nRows * nCols;
        int kernelSize = kRows * kCols;
        int fftSize = fftRows * fftCols;
        int cfftSize = fftRows * (fftCols / 2 + 1);

        // initilize basic one kernels and basic signal maps
        float *h_one_kernels = NULL;
        float *h_basic_signals = NULL;
        h_one_kernels = (float*)malloc(sizeof(float) * fftSize * kernelSize);
        h_basic_signals = (float*)malloc(sizeof(float) * fftSize * kernelSize);
        memset(h_one_kernels, 0, sizeof(float) * fftSize * kernelSize);
        memset(h_basic_signals, 0, sizeof(float) * fftSize * kernelSize);

        for(int i = 0; i < kRows; i++)
            for(int j = 0; j < kCols; j++){
                int kerIdx = i * kCols + j;
                int oneIdx = i * fftCols + j;
                h_one_kernels[kerIdx * fftSize + oneIdx] = 1.0;
        }
#if DEBUG
        for(int i = 0; i < kernelSize; i++){
            for(int j = 0; j < fftRows; j++){
        	for(int k = 0; k < fftCols; k++)
                    printf("%f ", h_one_kernels[i * fftSize + j * fftCols + k]);
                printf("\n");
            }
            printf("\n");
        }
#endif
        float *d_one_kernels = NULL;
        hipMalloc((void**)&d_one_kernels, sizeof(float) * fftSize * kernelSize);
        hipMemcpy(d_one_kernels, h_one_kernels, sizeof(float) * fftSize * kernelSize, hipMemcpyHostToDevice);

        // device memory pointer
        float *d_signal = NULL;   // device storage of input signal
        float *d_paddedSignal = NULL;   // device storage of input signal
        float *d_basic_signals = NULL;   // device storage of input signal
        float *d_results = NULL;
        
        hipMalloc((void**)&d_signal, sizeof(float) * dataSize);
        hipMalloc((void**)&d_paddedSignal, sizeof(float) * fftSize);
        //hipMalloc((void**)&d_basic_signals, sizeof(float) * fftSize * kernelSize);
        hipMalloc((void**)&d_results, sizeof(float) * fftSize * kernelCount);

        // timer.restart();
        hipMemset(d_signal, 0, dataSize * sizeof(float));
        hipMemset(d_paddedSignal, 0, fftSize * sizeof(float));
        //hipMemset(d_basic_signals, 0, fftSize * kernelSize * sizeof(float));
        hipMemset(d_results, 0, fftSize * kernelCount * sizeof(float));
        // timerMap["memset"].push_back(timer.elapsed());

        //for(int i = 0;i<10;i++){
        //    printf("%f\n", paddedSignal[i]);
        //    printf("%f\n", paddedKernels[i]);
        //}
        // timer.restart();
        hipMemcpy(d_signal, Signal, dataSize * sizeof(float), hipMemcpyHostToDevice);
        // padded signal
        dim3 threadBlock(16, 16, 4);
        dim3 BlockGrid(iDivUp(fftCols, threadBlock.x), iDivUp(fftRows, threadBlock.y), 1);
        padDataWithZeros<<<BlockGrid, threadBlock>>>(d_paddedSignal, d_signal, fftCols, fftRows, nCols, nRows, 1);      

#if DEBUG
        float *h_paddedSignal = (float*)malloc(sizeof(float) * fftSize);
        hipMemcpy(h_paddedSignal, d_paddedSignal, fftSize * sizeof(float), hipMemcpyDeviceToHost);
        for(int i = 0;i<fftRows;i++){
            for(int j = 0;j<fftCols;j++)
            	printf("%f ", h_paddedSignal[i * fftCols + j]);
            printf("\n");
	}
        printf("\n");
        // timerMap["memcpy"].push_back(timer.elapsed());
#endif

        hipfftComplex *d_signal_fft;
        hipfftComplex *d_one_kernels_fft;

        hipMalloc((void**)&d_signal_fft, cfftSize * sizeof(hipfftComplex));
        hipMalloc((void**)&d_one_kernels_fft, cfftSize * kernelSize * sizeof(hipfftComplex));
        //hipMemset(d_signal_fft, 0, cfftSize * sizeof(hipfftComplex));
        //hipMemset(d_one_kernels_fft, 0, cfftSize * kernelSize * sizeof(hipfftComplex));

        // for hipfftPlanMany parameters
        int idist = fftRows * fftCols;
        int odist = fftRows * (fftCols / 2 + 1);
        int inembed[] = {fftRows, fftCols};
        int onembed[] = {fftRows, fftCols / 2 + 1};
        int n[2] = {fftRows, fftCols};

        hipfftPlan2d(&fwplan_signal, fftRows, fftCols, HIPFFT_R2C);
        hipfftPlanMany(&fwplan_kernels, 2, n, inembed, 1, idist, onembed, 1, odist, HIPFFT_R2C, kernelSize);
        hipfftPlanMany(&bwplan, 2, n, onembed, 1, odist, inembed, 1, idist, HIPFFT_C2R, kernelSize);
        //hipfftPlan1d(&fwplan_signal, nCols, HIPFFT_R2C, 1);
        //hipfftPlan1d(&fwplan_kernels, nCols, HIPFFT_R2C, kernelCount);
        //hipfftPlan1d(&bwplan, nCols, HIPFFT_C2R, kernelCount);

        // timer.restart();
        //create_plan(fwplan_signal, real_size, HIPFFT_Z2Z, 1);
        //create_plan(fwplan_kernels, real_size, HIPFFT_Z2Z, kernelCount);
        //create_plan(bwplan, real_size, HIPFFT_Z2Z, kernelCount);
        // timerMap["create_plans"].push_back(timer.elapsed());

        // timer.restart();
        hipfftExecR2C(fwplan_signal, d_paddedSignal, d_signal_fft);
        hipfftExecR2C(fwplan_kernels, d_one_kernels, d_one_kernels_fft);
        // timerMap["forward"].push_back(timer.elapsed());

#if DEBUG
        hipfftComplex *h_one_kernels_fft = (hipfftComplex*)malloc(sizeof(hipfftComplex) * kernelSize * cfftSize);
        hipMemcpy(h_one_kernels_fft, d_one_kernels_fft, sizeof(hipfftComplex) * kernelSize * cfftSize, hipMemcpyDeviceToHost);
        for(int k = 0 ; k < kernelSize; k++){
            for (int i = 0 ; i < cfftSize; i++)
                printf("%.2f+%.2fj ", h_one_kernels_fft[k * cfftSize + i].x, h_one_kernels_fft[k * cfftSize + i].y);
            printf("\n");
        }
        printf("\n");
#endif

        // timer.restart();
        thrust::device_ptr<hipfftComplex> thr_signal_fft(d_signal_fft);
        for(int i = 0 ;i < kernelSize; i++){
            thrust::device_ptr<hipfftComplex> thr_one_kernel_fft(d_one_kernels_fft + i * odist);
            thrust::transform(thr_signal_fft, thr_signal_fft + odist, thr_one_kernel_fft, thr_one_kernel_fft,
                              complexMultiplier(1.0 / idist));
            // timerMap["multiply"].push_back(timer.elapsed());
        }

        // timer.restart();
        //hipfftExecC2R(bwplan, d_one_kernels_fft, d_basic_signals);
        hipfftExecC2R(bwplan, d_one_kernels_fft, d_one_kernels);
        d_basic_signals = d_one_kernels;

#if DEBUG
        hipMemcpy(h_basic_signals, d_basic_signals, kernelSize * fftSize * sizeof(float), hipMemcpyDeviceToHost);
        for (int i = 0 ; i < kernelSize;i++){
	    for(int r = 0 ; r < fftRows; r++){
		for(int c = 0; c< fftCols;c++)
		    printf("%f ", h_basic_signals[i * fftSize + r * fftCols + c]);
                printf("\n");
	    }
	    printf("\n");
        }
	printf("\n");
#endif
        //hipfftExecZ2Z(*bwplan, d_kernels_fft, d_kernels, HIPFFT_BACKWARD);
        // timerMap["backward"].push_back(timer.elapsed());

        // timer.restart();
        // hipMemcpy(h_basic_signals, d_one_kernels_fft, kernelSize * idist * sizeof(float), hipMemcpyDeviceToHost);
       
        // timerMap["copy_back"].push_back(timer.elapsed());
        // timerMap["total_time"].push_back(globalTimer.elapsed());

        
        //for (int r = 0 ; r < fftRows; r++)
        //    for(int c = 0; c < fftCols; c++){
        //	for(int j = 0 ; j < kernelSize; j++){
        //            float a1 = results[r * fftCols + c];
        //            float a2 = h_basic_signals[j * fftSize + r * fftCols + c];
	//    	    results[r * fftCols + c] += h_basic_signals[j * fftSize + r * fftCols + c];
        //            printf("%f+%f=%f ", a1, a2, results[r * fftCols + c]);
	//	}
	//    printf("\n");
        //}
        //for(int i = 0; i < kernelCount; i++){
        //    for(int j = 0 ; j < kernelSize; j++){
        //        for (int r = 0 ; r < fftRows; r++)
        //            for(int c = 0; c < fftCols; c++)        
	//		results[i * fftSize + r * fftCols + c] += Kernels[i * kernelSize + j] * h_basic_signals[j * fftSize + r * fftCols + c];
        //    }
        //}


        for(int i = 0; i < kernelCount; i++){
            thrust::device_ptr<float> thr_results(d_results + i * fftSize);
            for(int j = 0 ; j < kernelSize; j++){
                thrust::device_ptr<float> thr_basic_signals(d_basic_signals + j * fftSize);
		saxpy_fast(Kernels[i * kernelSize + j], thr_basic_signals, thr_results, fftSize);
            }
        }

        hipMemcpy(results, d_results, kernelCount * fftSize * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_one_kernels);
        hipFree(d_signal);
        hipFree(d_paddedSignal);
        hipFree(d_results);
        hipFree(d_signal_fft);
        hipFree(d_one_kernels_fft);

    }

    void convolve_turbo_fft_v2(float * Signal, float * Kernels, int kernelCount, 
                             int nRows, int nCols, int kRows, int kCols, float * results)
    {

        hipfftHandle fwplan_signal, fwplan_kernels, bwplan; 
        // timer timer, globalTimer;
        // globalTimer.restart();

        int fftRows = nRows + kRows - 1;
        int fftCols = nCols + kCols - 1;
        int dataSize = nRows * nCols;
        int kernelSize = kRows * kCols;
        int fftSize = fftRows * fftCols;
        int cfftSize = fftRows * fftCols;
        int group_num = kernelCount / 2;

	// parameter for hipfftPlanMany
        int idist = fftRows * fftCols;
        int odist = fftRows * fftCols;
        int inembed[] = {fftRows, fftCols};
        int onembed[] = {fftRows, fftCols};
        int n[2] = {fftRows, fftCols};

#if DEBUG
        for(int i = 0;i<nRows;i++){
            for(int j = 0;j<nCols;j++)
                printf("%f ", Signal[i * nCols + j]);
            printf("\n");
        }
        printf("\n");
        for(int i = 0;i<kRows;i++){
            for(int j = 0;j<kCols;j++)
                printf("%f ", Kernels[i * kCols + j]);
            printf("\n");
	}
        printf("\n");
#endif

        // device memory pointer
        float *d_signal = NULL;   // device storage of input signal
        float *d_kernels = NULL;  // device storage of kernels and final results
        hipfftComplex *d_paddedSignal = NULL;
        hipfftComplex *d_paddedKernels = NULL;  // device storage of kernels and final results
        float *d_results = NULL;
        hipfftComplex *d_signal_fft;
        hipfftComplex *d_kernels_fft;
        hipfftComplex *d_w_results;
        
        hipMalloc((void**)&d_signal, sizeof(float) * dataSize);
        hipMalloc((void**)&d_kernels, sizeof(float) * kernelSize * kernelCount);
        hipMalloc((void**)&d_paddedSignal, sizeof(hipfftComplex) * fftSize);
        hipMalloc((void**)&d_paddedKernels, sizeof(hipfftComplex) * fftSize * group_num);
        hipMalloc((void**)&d_results, sizeof(float) * fftSize * kernelCount);
        hipMalloc((void**)&d_signal_fft, cfftSize * sizeof(hipfftComplex));
        hipMalloc((void**)&d_kernels_fft, cfftSize * group_num * sizeof(hipfftComplex));
        hipMalloc((void**)&d_w_results, cfftSize * group_num * sizeof(hipfftComplex));

        // timer.restart();
        hipMemset(d_signal, 0, dataSize * sizeof(float));
        hipMemset(d_kernels, 0, kernelSize * kernelCount * sizeof(float));
        hipMemset(d_paddedSignal, 0, fftSize * sizeof(hipfftComplex));
        hipMemset(d_paddedKernels, 0, fftSize * group_num * sizeof(hipfftComplex));
        hipMemset(d_results, 0, fftSize * kernelCount * sizeof(float));
        // timerMap["memset"].push_back(timer.elapsed());

        // timer.restart();
        hipMemcpy(d_signal, Signal, dataSize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_kernels, Kernels, kernelSize * kernelCount * sizeof(float), hipMemcpyHostToDevice);
        //hipMemcpy(Kernels, d_kernels, kernelSize * kernelCount * sizeof(float), hipMemcpyDeviceToHost);
        //for(int i = 0;i<kernelSize;i++)
        //    printf("%f\n", Kernels[i]);
        //printf("\n");
        // timerMap["memcpy"].push_back(timer.elapsed());

        // padded signal
        dim3 threadBlock(16, 16, 4);
        dim3 dataBlockGrid(iDivUp(fftCols, threadBlock.x), iDivUp(fftRows, threadBlock.y), 1);
        dim3 kernelBlockGrid(iDivUp(fftCols, threadBlock.x), iDivUp(fftRows, threadBlock.y), iDivUp(group_num, threadBlock.z));
        //dim3 kernelBlockGrid(iDivUp(kCols, threadBlock.x), iDivUp(kRows, threadBlock.y), iDivUp(group_num, threadBlock.z));
        //dim3 kernelBlockGrid(1, 1, iDivUp(group_num, threadBlock.z));
	    int padding_time = 0;
        auto start_pad = std::chrono::steady_clock::now();

        padDataWithZerosComplex<<<dataBlockGrid, threadBlock>>>(d_paddedSignal, d_signal, fftCols, fftRows, nCols, nRows, 1); 
        padKernelWithZerosComplex<<<kernelBlockGrid, threadBlock>>>(d_paddedKernels, d_kernels, fftCols, fftRows, kCols, kRows, group_num);
        hipDeviceSynchronize();

        auto end_pad = std::chrono::steady_clock::now();
        padding_time += static_cast<int>(std::chrono::duration<double, std::micro>(end_pad - start_pad).count() );
	//printf("padding time for %d group(s): %d.\n", group_num, padding_time);

#if DEBUG
        CudaCheckError();
        hipfftComplex *h_paddedSignal = (hipfftComplex*)malloc(sizeof(hipfftComplex) * fftSize);
        hipfftComplex *h_paddedKernels = (hipfftComplex*)malloc(sizeof(hipfftComplex) * fftSize * group_num);

        hipMemcpy(h_paddedSignal, d_paddedSignal, sizeof(hipfftComplex) * fftSize, hipMemcpyDeviceToHost);
        hipMemcpy(h_paddedKernels, d_paddedKernels, sizeof(hipfftComplex) * fftSize * group_num, hipMemcpyDeviceToHost);

        for(int i = 0;i<fftRows;i++){
            for(int j = 0;j<fftCols;j++)
            	printf("%f+%fi ", h_paddedSignal[i * fftCols + j].x, h_paddedSignal[i * fftCols + j].y);
            printf("\n");
	}
        printf("\n");
        for(int i = 0;i<fftRows;i++){
            for(int j = 0;j<fftCols;j++)
            	printf("%f+%fi ", h_paddedKernels[i * fftCols + j].x, h_paddedKernels[i * fftCols + j].y);
            printf("\n");
	}
        printf("\n");
		getchar();
#endif
        //hipMemset(d_signal_fft, 0, odist * sizeof(hipfftComplex));
        //hipMemset(d_kernels_fft, 0, odist * kernelCount * sizeof(hipfftComplex));

        hipfftPlan2d(&fwplan_signal, fftRows, fftCols, HIPFFT_C2C);
        hipfftPlanMany(&fwplan_kernels, 2, n, inembed, 1, idist, onembed, 1, odist, HIPFFT_C2C, group_num);
        hipfftPlanMany(&bwplan, 2, n, onembed, 1, odist, inembed, 1, idist, HIPFFT_C2C, group_num);
        //hipfftPlan1d(&fwplan_signal, nCols, HIPFFT_R2C, 1);
        //hipfftPlan1d(&fwplan_kernels, nCols, HIPFFT_R2C, kernelCount);
        //hipfftPlan1d(&bwplan, nCols, HIPFFT_C2R, kernelCount);

        // timer.restart();
        //create_plan(fwplan_signal, real_size, HIPFFT_Z2Z, 1);
        //create_plan(fwplan_kernels, real_size, HIPFFT_Z2Z, kernelCount);
        //create_plan(bwplan, real_size, HIPFFT_Z2Z, kernelCount);
        // timerMap["create_plans"].push_back(timer.elapsed());

        // timer.restart();

	// warm up
	{

            hipfftExecC2C(fwplan_signal, d_paddedSignal, d_signal_fft, HIPFFT_FORWARD);
            hipfftExecC2C(fwplan_kernels, d_paddedKernels, d_kernels_fft, HIPFFT_FORWARD);
            // timerMap["forward"].push_back(timer.elapsed());

            // timer.restart();

            dim3 dotBlock(32, 32);
            dim3 dotGrid(fftCols / 32, fftCols / 32, 8);
	    dotProduct<<<dotGrid, dotBlock>>>(d_signal_fft, fftCols, fftRows, 1.0 / fftSize, group_num/8, d_kernels_fft);
            hipDeviceSynchronize();

            // thrust::device_ptr<hipfftComplex> thr_signal_fft(d_signal_fft);
            // thrust::device_ptr<hipfftComplex> thr_kernel_fft(d_kernels_fft);
            // //thrust::transform(thr_signal_fft, thr_signal_fft + group_num * cfftSize, thr_kernel_fft, thr_kernel_fft, complexMultiplier(1.0 / fftSize));
            // for(int i = 0 ;i < group_num; i++){
            //     thrust::transform(thr_signal_fft, thr_signal_fft + cfftSize, thr_kernel_fft + i * cfftSize, thr_kernel_fft + i * cfftSize,
            //                       complexMultiplier(1.0 / fftSize));
            //     // timerMap["multiply"].push_back(timer.elapsed());
            // }

#if VERBOSE
            size_t freeMem, totalMem;
            hipMemGetInfo(&freeMem, &totalMem);
            std::cout << "Free: " << freeMem << ", Total: " << totalMem << std::endl;
#endif

            //hipMalloc((void**)&d_conj_w_results, cfftSize * group_num * sizeof(hipfftComplex));
            // timer.restart();
            hipfftExecC2C(bwplan, d_kernels_fft, d_w_results, HIPFFT_BACKWARD);
	}


        hipDeviceSynchronize();

	// timing record
	int fft_time = 0;
	int dot_product_time = 0;
	int ifft_time = 0;

        int num_repeats = NUM_ITERATIONS;
		//hipfftComplex* d_double_signal_fft;
		//int duplicate = 16;
        //hipMalloc((void**)&d_double_signal_fft, fftSize * duplicate * sizeof(hipfftComplex));
        for (int i = 0; i < num_repeats; ++i) 
	{

            auto start = std::chrono::steady_clock::now();
            hipfftExecC2C(fwplan_signal, d_paddedSignal, d_signal_fft, HIPFFT_FORWARD);
            hipfftExecC2C(fwplan_kernels, d_paddedKernels, d_kernels_fft, HIPFFT_FORWARD);
            hipDeviceSynchronize();
            auto end = std::chrono::steady_clock::now();
            fft_time += static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() );

            // timerMap["forward"].push_back(timer.elapsed());

            // timer.restart();

	    start = std::chrono::steady_clock::now();

		    //// thrust implementation
			//for(int c = 0; c < duplicate;c++)
            //    hipMemcpy(d_double_signal_fft + c * fftSize, d_signal_fft, sizeof(hipfftComplex) * fftSize, hipMemcpyDeviceToDevice);
            //thrust::device_ptr<hipfftComplex> thr_signal_fft(d_double_signal_fft);
            //thrust::device_ptr<hipfftComplex> thr_kernel_fft(d_kernels_fft);
            ////thrust::transform(thr_signal_fft, thr_signal_fft + group_num * cfftSize, thr_kernel_fft, thr_kernel_fft, complexMultiplier(1.0 / fftSize));
            //for(int i = 0 ;i < group_num / duplicate; i++){
            //    thrust::transform(thr_signal_fft, thr_signal_fft + duplicate * fftSize, thr_kernel_fft + i * duplicate * fftSize, thr_kernel_fft + i * duplicate * fftSize,
            //                      complexMultiplier(1.0 / fftSize));
            //    // timerMap["multiply"].push_back(timer.elapsed());
            //}
            dim3 dotBlock(32, 32);
            dim3 dotGrid(fftCols / 32, fftCols / 32, 8);
			dotProduct<<<dotGrid, dotBlock>>>(d_signal_fft, fftCols, fftRows, 1.0 / fftSize, group_num/8, d_kernels_fft);
            hipDeviceSynchronize();
	    end = std::chrono::steady_clock::now();
            dot_product_time += static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() );

#if VERBOSE
            size_t freeMem, totalMem;
            hipMemGetInfo(&freeMem, &totalMem);
            std::cout << "Free: " << freeMem << ", Total: " << totalMem << std::endl;
#endif

            //hipMalloc((void**)&d_conj_w_results, cfftSize * group_num * sizeof(hipfftComplex));
            // timer.restart();
	    start = std::chrono::steady_clock::now();
            hipfftExecC2C(bwplan, d_kernels_fft, d_w_results, HIPFFT_BACKWARD);
            hipDeviceSynchronize();
	    end = std::chrono::steady_clock::now();
            ifft_time += static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() );

	}

        int avg_fft_time = fft_time / num_repeats;
        int avg_dot_product_time = dot_product_time / num_repeats;
        int avg_ifft_time = ifft_time / num_repeats;
	int avg_total_time = avg_fft_time + avg_dot_product_time + avg_ifft_time;
	printf("turbo_fft_gpu_v2 total time:%d\n", avg_total_time);

        // split complex results
        SplitComplex<<<kernelBlockGrid, threadBlock>>>(d_results, d_w_results, fftCols, fftRows, group_num);
        //ConjComplex<<<kernelBlockGrid, threadBlock>>>(d_conj_w_results, d_w_results, fftCols, fftRows, group_num);
        //hipfftExecZ2Z(*bwplan, d_kernels_fft, d_kernels, HIPFFT_BACKWARD);
        // timerMap["backward"].push_back(timer.elapsed());

        // timer.restart();
        hipMemcpy(results, d_results, kernelCount * fftSize * sizeof(float), hipMemcpyDeviceToHost);
       
        // timerMap["copy_back"].push_back(timer.elapsed());
        // timerMap["total_time"].push_back(globalTimer.elapsed());

        // free GPU memory
        hipFree(d_signal);
        hipFree(d_kernels);
        hipFree(d_paddedSignal);
        hipFree(d_paddedKernels);
        hipFree(d_results);
        hipFree(d_signal_fft);
        hipFree(d_kernels_fft);
        hipFree(d_w_results);

    }

    // data and qdVectors are in GPUs
    void qdVector_construct(const hipfftComplex* d_data, int group_num, int N, int kCols, hipfftComplex* d_qdVectors){

	int fftSize = N * N;

	bool use_gpu = 1;
	bool sparse = 1;
	if (use_gpu){
	    if(sparse){
	        // gpu sparse version of qdVector construction
                dim3 threadBlock(32, 1);
                dim3 BlockGrid(group_num, iDivUp(non_empty_qdV, 32));
	        //printf("group num: %d, qdV num: %d.\n", group_num, non_empty_qdV);
	        //printf("%d, %d, %d, %d.\n", threadBlock.x, threadBlock.y, BlockGrid.x, BlockGrid.y);
                qdVectorConstruct_sparse<<<BlockGrid, threadBlock>>>(d_data, d_sparse_input_xs, d_sparse_input_ys, d_begin_arr, d_end_arr, d_qdV_arr, N, kCols, group_num, non_empty_qdV, d_qdVectors); 
                hipDeviceSynchronize();

/*
	        hipfftComplex *h_qdVectors = (hipfftComplex*)malloc(sizeof(hipfftComplex) * fftSize * 3 / 2 * group_num);
	        hipMemcpy(h_qdVectors, d_qdVectors, sizeof(hipfftComplex) * fftSize * 3 / 2 * group_num, hipMemcpyDeviceToHost);
                for(int i = 0;i< N * 3 / 2 ;i++){
	            printf("(%d %d): ", p[i], s[i]);
                    for(int j = 0;j<N;j++)
                    	printf("%f+%fi ", h_qdVectors[i * N + j].x, h_qdVectors[i * N + j].y);
                    printf("\n");
	            getchar();
	        }
                printf("\n");
*/

	    }
	    else{
	        // gpu version of qdVector construction
	        int qdV_num = N * 3 / 2;
                dim3 threadBlock(32, N / 32);
                dim3 BlockGrid(group_num, qdV_num);
	        printf("group num: %d, qdV num: %d.\n", group_num, qdV_num);
	        printf("%d, %d, %d, %d.\n", threadBlock.x, threadBlock.y, BlockGrid.x, BlockGrid.y);
                qdVectorConstruct<<<BlockGrid, threadBlock>>>(d_data, d_input_xs, d_input_ys, N, group_num, qdV_num, d_qdVectors); 
                hipDeviceSynchronize();


	        hipfftComplex *h_qdVectors = (hipfftComplex*)malloc(sizeof(hipfftComplex) * fftSize * 3 / 2 * group_num);
	        hipMemcpy(h_qdVectors, d_qdVectors, sizeof(hipfftComplex) * fftSize * 3 / 2 * group_num, hipMemcpyDeviceToHost);
                for(int i = 0;i< N * 3 / 2 ;i++){
	            printf("(%d %d): ", p[i], s[i]);
                    for(int j = 0;j<N;j++)
                    	printf("%f+%fi ", h_qdVectors[i * N + j].x, h_qdVectors[i * N + j].y);
                    printf("\n");
	            getchar();
	        }
                printf("\n");

	    }
	}
	else
	// cpu version of qdVector construction
	{
            hipfftComplex *h_paddedKernels = (hipfftComplex*)malloc(sizeof(hipfftComplex) * fftSize * group_num);  // padded kernels
            hipMemcpy(h_paddedKernels, d_data, sizeof(hipfftComplex) * fftSize * group_num, hipMemcpyDeviceToHost);
	    hipfftComplex *h_qdVectors = (hipfftComplex*)malloc(sizeof(hipfftComplex) * fftSize * 3 / 2 * group_num);
            memset(h_qdVectors, 0, sizeof(hipfftComplex) * fftSize * 3 / 2 * group_num);

	    if (sparse){

	        // construct 1d vectors for 1d-fft
	        for(int k = 0; k < group_num;k++){
	            int kOffset = k * fftSize * 3 / 2;
	            for (int q = 0; q < non_empty_qdV; q++){
	                for (int s = begin_arr[q] ; s < end_arr[q];s++){
	                    int idx_x = sparse_input_xs[s];
	                    int idx_y = sparse_input_ys[s];
	                    h_qdVectors[kOffset + qdV_arr[q]].x += h_paddedKernels[k * fftSize + idx_x * N + idx_y].x;
	                    h_qdVectors[kOffset + qdV_arr[q]].y += h_paddedKernels[k * fftSize + idx_x * N + idx_y].y;
		        }
		    }
		}
	    
	    }
	    else
	    {
#if DEBUG
	        printf("group_num: %d.\n", group_num);
                for(int i = 0;i<N;i++){
                    for(int j = 0;j<N;j++)
                    	printf("%f+%fi ", h_paddedKernels[i * N + j].x, h_paddedKernels[i * N + j].y);
                    printf("\n");
	        }
                printf("\n");
#endif

	        // construct 1d vectors for 1d-fft
	        for(int k = 0; k < group_num;k++){
	            int kOffset = k * fftSize * 3 / 2;
                    for(int i = 0; i < N + N/2;i++)
	                for(int t = 0; t < N; t++)
	                    for(int j = 0; j < N;j++){
	            	        int offset = i * fftSize + t * N + j;
	            	        int idx_x = input_xs[offset];
	            	        int idx_y = input_ys[offset];
	                        h_qdVectors[kOffset + i * N + t].x += h_paddedKernels[k * fftSize + idx_x * N + idx_y].x;
	                        h_qdVectors[kOffset + i * N + t].y += h_paddedKernels[k * fftSize + idx_x * N + idx_y].y;
	            	}
	        }

	    }

	    hipMemcpy(d_qdVectors, h_qdVectors, sizeof(hipfftComplex) * fftSize * 3 / 2 * group_num, hipMemcpyHostToDevice);

#if DEBUG
        CudaCheckError();
        for(int i = 0;i<N * 3 / 2 ;i++){
	    printf("(%d %d):", p[i], s[i]);
            for(int j = 0;j<N;j++)
            	printf("%f+%fi ", h_qdVectors[i * N + j].x, h_qdVectors[i * N + j].y);
            printf("\n");
	    getchar();
	}
        printf("\n");
        printf("finish vector construction.\n");
#endif
	}


    }


    // fast fft with QD algorithm, the input signals and output results are on GPUs.
    void fast_fft(hipfftComplex * d_kernels, int group_num, int fftCols, int fftRows, int kCols, int kRows, hipfftComplex * d_results, bool forward = true)
    {

	int fftSize = fftCols * fftRows;

	// initialize space for qdVectors and ifft_results on GPUs
	if(d_qdVectors == NULL){

#if DEBUG
	    printf("Allocate gpu space of qdVectors...\n");
#endif
            hipMalloc((void**)&d_qdVectors, sizeof(hipfftComplex) * fftSize * 3 / 2 * group_num);
            hipMalloc((void**)&d_w_results, sizeof(hipfftComplex) * fftSize * 3 / 2 * group_num);
	}
	if (p == NULL){
#if DEBUG
	    printf("Construct index array for qd-fft for the first time...\n");
#endif
	    // initilize p and s
	    p = new int[fftCols + fftCols / 2];
	    s = new int[fftCols + fftCols / 2];
	    for(int i = 0 ; i < fftCols; i ++){
	        p[i] = 1;
	        s[i] = i;
	    }
	    for(int i = fftCols; i < fftCols + fftCols / 2;i++){
	        p[i] = (i - fftCols) * 2;
	        s[i] = 1;
	    }

#if DEBUG
            printf("finish initilization of (p, s).\n");
	    for(int i = 0 ; i < fftCols + fftCols / 2;i++)
	        printf("%d: p:%d, s:%d.\n", i, p[i], s[i]);
#endif

	    // initilize indices of get data, 1.5 * N * 2N^2, indices of out data, 
	    int N = fftCols;
	    input_xs = new int[N * 3 / 2 * (N * N)];
	    input_ys = new int[N * 3 / 2 * (N * N)];
	    output_xs = new int[N * 3 / 2 * N];
	    output_ys = new int[N * 3 / 2 * N];

            for(int i = 0; i < fftCols + fftCols/2;i++){
	        for(int t = 0 ; t < fftCols;t++){
	    	output_xs[i * fftCols + t] = (t * p[i]) % fftCols;
	    	output_ys[i * fftCols + t] = (t * s[i]) % fftCols;
	        }
	    }

	    for(int i = 0; i < fftCols + fftCols/2;i++)
	        for(int t = 0; t < fftCols; t++){
	    	int offset = i * fftSize + t * fftCols;
	            find_input_index(p[i], s[i], t, fftCols, input_xs + offset, input_ys + offset);
            }
#if DEBUG
            printf("finish indices search.\n");
	    for(int i = 0; i < fftCols + fftCols/2;i++)
	        for(int t = 0; t < fftCols; t++){
	    	if(p[i] != 18 || s[i] != 1)
	    		continue;
	    	printf("(%d, %d, %d):", p[i], s[i], t);
	    	for (int j = 0 ;j < fftCols;j++)
	    	    printf("[%d, %d]", input_xs[i * fftSize + t * fftCols + j], input_ys[i * fftSize + t * fftCols + j]);
	    	printf("\n");
	    	getchar();
	        }
#endif

	    if(sparse_input_xs == NULL){
	        non_zero = 0;
	        non_empty_qdV = 0;
	        for(int i = 0; i < fftCols + fftCols/2;i++)
	            for(int t = 0; t < fftCols; t++){
	                int tmp_non_zero = 0;
	                for(int j = 0; j < fftCols;j++)
	            	if(input_xs[i * fftSize + t * fftCols + j] < kCols && input_ys[i * fftSize + t * fftCols + j] < kCols)
	            	    tmp_non_zero++;

	                if(tmp_non_zero != 0){
	            	    non_empty_qdV++;
	            	    non_zero += tmp_non_zero;
	                }
	            }
	            
#if DEBUG
		printf("non_zero: %d, non_empty_qdV: %d.\n", non_zero, non_empty_qdV);
#endif
    	        sparse_input_xs = new int[non_zero];
    	        sparse_input_ys = new int[non_zero];
    	        begin_arr = new int[non_empty_qdV];
    	        end_arr = new int[non_empty_qdV];
    	        qdV_arr = new int[non_empty_qdV];

		int start_idx = 0;
		int end_idx = 0;
		int qdV_idx = 0;
	        for(int i = 0; i < fftCols + fftCols/2;i++)
	            for(int t = 0; t < fftCols; t++){
	                for(int j = 0; j < fftCols;j++)
	            	    if(input_xs[i * fftSize + t * fftCols + j] < kCols && input_ys[i * fftSize + t * fftCols + j] < kCols)
			    {
				sparse_input_xs[end_idx] = input_xs[i * fftSize + t * fftCols + j];
				sparse_input_ys[end_idx] = input_ys[i * fftSize + t * fftCols + j];
				end_idx++;
			    }

	                if(end_idx != start_idx){
	            	    begin_arr[qdV_idx] = start_idx;
	            	    end_arr[qdV_idx] = end_idx;
			    start_idx = end_idx;
			    qdV_arr[qdV_idx] = i * fftCols + t;
			    qdV_idx++;
	                }
	            }

		//// print out for check
	        //for (int q = 0; q < non_empty_qdV; q++){
	        //    printf("%d[%d]: ", q, qdV_arr[q]);
	        //    for (int s = begin_arr[q] ; s < end_arr[q];s++)
	        //        printf("(%d, %d) ", sparse_input_xs[s], sparse_input_ys[s]);
	        //    printf("\n");
		//    getchar();
	        //}
	    }
	    if(d_input_xs == NULL){
		// initialize GPU index array
                hipMalloc((void**)&d_input_xs, sizeof(int) * N * 3 / 2 * fftSize);
                hipMalloc((void**)&d_input_ys, sizeof(int) * N * 3 / 2 * fftSize);
                hipMalloc((void**)&d_output_xs, sizeof(int) * N * 3 / 2 * N);
                hipMalloc((void**)&d_output_ys, sizeof(int) * N * 3 / 2 * N);
		
	        hipMemcpy(d_input_xs, input_xs, sizeof(int) * N * 3 / 2 * fftSize, hipMemcpyHostToDevice);
	        hipMemcpy(d_input_ys, input_ys, sizeof(int) * N * 3 / 2 * fftSize, hipMemcpyHostToDevice);
	        hipMemcpy(d_output_xs, output_xs, sizeof(int) * N * 3 / 2 * N, hipMemcpyHostToDevice);
	        hipMemcpy(d_output_ys, output_ys, sizeof(int) * N * 3 / 2 * N, hipMemcpyHostToDevice);
	    }
	    if(d_sparse_input_xs == NULL){
		// initialize GPU index array
                hipMalloc((void**)&d_sparse_input_xs, sizeof(int) * non_zero);
                hipMalloc((void**)&d_sparse_input_ys, sizeof(int) * non_zero);
                hipMalloc((void**)&d_begin_arr, sizeof(int) * non_empty_qdV);
                hipMalloc((void**)&d_end_arr, sizeof(int) * non_empty_qdV);
                hipMalloc((void**)&d_qdV_arr, sizeof(int) * non_empty_qdV);

	        hipMemcpy(d_sparse_input_xs, sparse_input_xs, sizeof(int) * non_zero, hipMemcpyHostToDevice);
	        hipMemcpy(d_sparse_input_ys, sparse_input_ys, sizeof(int) * non_zero, hipMemcpyHostToDevice);
		hipMemcpy(d_begin_arr, begin_arr, sizeof(int) * non_empty_qdV, hipMemcpyHostToDevice);
		hipMemcpy(d_end_arr, end_arr, sizeof(int) * non_empty_qdV, hipMemcpyHostToDevice);
		hipMemcpy(d_qdV_arr, qdV_arr, sizeof(int) * non_empty_qdV, hipMemcpyHostToDevice);
	    } 

	}

	int vec_con_time = 0;
        auto start = std::chrono::steady_clock::now();
        qdVector_construct(d_kernels, group_num, fftCols, kCols, d_qdVectors);
	//hipMemcpy(d_qdVectors, h_qdVectors, sizeof(hipfftComplex) * fftSize * 3 / 2 * group_num, hipMemcpyHostToDevice);
        auto end = std::chrono::steady_clock::now();
        vec_con_time += static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() );

#if QDFFT_MEASURE
	printf("vec_construct:%d, ", vec_con_time);
#endif

	int fft_time = 0;
        start = std::chrono::steady_clock::now();
	if (forward)
            hipfftExecC2C(plan1d, d_qdVectors, d_w_results, HIPFFT_FORWARD);
	else
            hipfftExecC2C(plan1d, d_qdVectors, d_w_results, HIPFFT_BACKWARD);

        hipDeviceSynchronize();
        end = std::chrono::steady_clock::now();
        fft_time += static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() );

#if QDFFT_MEASURE
	printf("fft_time:%d, ", fft_time);
#endif

#if DEBUG
        printf("finish fft.\n");
#endif

	int use_cpu = 0;
	int re_arrange_time = 0;
        start = std::chrono::steady_clock::now();
	if (use_cpu)
	// cpu version of re-arrange data
	{
	    hipfftComplex* h_results = (hipfftComplex*)malloc(group_num * fftSize * sizeof(hipfftComplex));
	    hipfftComplex* h_w_results = (hipfftComplex*)malloc(group_num * fftSize * 3 / 2 * sizeof(hipfftComplex));
            hipMemcpy(h_w_results, d_w_results, group_num * fftSize * 3 / 2 * sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	    for(int k = 0; k < group_num;k++){
	        int kOffset = k * fftSize * 3 / 2;
                for(int i = 0; i < fftCols + fftCols/2;i++)
	            for(int t = 0; t < fftCols; t++){
	    	    int idx_x = output_xs[i * fftCols + t];
	    	    int idx_y = output_ys[i * fftCols + t];
	    	    h_results[k * fftSize + idx_x * fftCols + idx_y].x = h_w_results[kOffset + i * fftCols + t].x;
	    	    h_results[k * fftSize + idx_x * fftCols + idx_y].y = h_w_results[kOffset + i * fftCols + t].y;
	    	}
	    	
	    }

#if DEBUG
            CudaCheckError();
            for(int i = 0;i<fftRows;i++){
                for(int j = 0;j<fftCols;j++)
                	printf("%f+%fi ", h_results[i * fftCols + j].x, h_results[i * fftCols + j].y);
                printf("\n");
	    }
            printf("\n");
	    getchar();
#endif

	    hipMemcpy(d_results, h_results, sizeof(hipfftComplex) * fftSize * group_num, hipMemcpyHostToDevice);
	}
	else
	{
	    int N = fftCols;

	    ////global memory
            //dim3 threadBlock(32, 1);
            //dim3 BlockGrid(group_num, iDivUp(N * N * 3 / 2, 32));
            //qdVectorReconstruct<<<BlockGrid, threadBlock>>>(d_w_results, d_output_xs, d_output_ys, N, group_num, d_results); 

	    //shared memory
		    int rows = 4;
            //dim3 top_half_threadBlock(N, 4);
            //dim3 top_half_BlockGrid(group_num, N/4);
            //qdVectorReconstruct_Shared<<<top_half_BlockGrid, top_half_threadBlock>>>(d_w_results, d_output_xs, d_output_ys, N, group_num, d_results, true); 
            dim3 top_half_threadBlock(N, N/4);
            dim3 top_half_BlockGrid(group_num, 1);
            qdVectorReconstruct_Small<<<top_half_BlockGrid, top_half_threadBlock>>>(d_w_results, d_output_xs, d_output_ys, N, group_num, d_results, true); 
            //dim3 threadBlock(N / 2, 1);
            //dim3 BlockGrid(group_num, 1);
            //qdVectorReconstruct_Shared<<<BlockGrid, threadBlock>>>(d_w_results, d_output_xs, d_output_ys, N, group_num, d_results, false); 
            hipDeviceSynchronize();

	
	}
        end = std::chrono::steady_clock::now();
        re_arrange_time += static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() );

#if QDFFT_MEASURE
	printf("data_rearrange:%d.\n", re_arrange_time);
#endif

    }

    void convolve_turbo_fft_v3(float * Signal, float * Kernels, int kernelCount, 
                             int nRows, int nCols, int kRows, int kCols, float * results)
    {

        hipfftHandle fwplan_signal, fwplan_kernels, bwplan; 
        // timer timer, globalTimer;
        // globalTimer.restart();

        int fftRows = nRows + kRows - 1;
        int fftCols = nCols + kCols - 1;
        int dataSize = nRows * nCols;
        int kernelSize = kRows * kCols;
        int fftSize = fftRows * fftCols;
        int cfftSize = fftRows * fftCols;
        int group_num = kernelCount / 2;

	// parameters for cufftPlanMany1d
        int idist = fftCols;
        int odist = fftCols;
        int inembed[] = {0};
        int onembed[] = {0};
        int n[1] = {fftCols};

	// parameter for cufftPlanMany2d
        int idist_2d = fftRows * fftCols;
        int odist_2d = fftRows * fftCols;
        int inembed_2d[] = {fftRows, fftCols};
        int onembed_2d[] = {fftRows, fftCols};
        int n_2d[2] = {fftRows, fftCols};

#if DEBUG
        for(int i = 0;i<nRows;i++){
            for(int j = 0;j<nCols;j++)
                printf("%f ", Signal[i * nCols + j]);
            printf("\n");
        }
        printf("\n");
        for(int i = 0;i<kRows;i++){
            for(int j = 0;j<kCols;j++)
                printf("%f ", Kernels[i * kCols + j]);
            printf("\n");
	}
        printf("\n");
#endif

        // device memory pointer
        float *d_signal = NULL;   // device storage of input signal
        float *d_kernels = NULL;  // device storage of kernels and final results
        hipfftComplex *d_paddedSignal = NULL;
        hipfftComplex *d_paddedKernels = NULL;  // device storage of kernels and final results
        float *d_results = NULL;
        hipfftComplex *d_signal_fft;
        hipfftComplex *d_kernels_fft;
        hipfftComplex *d_w_results;
        
        hipMalloc((void**)&d_signal, sizeof(float) * dataSize);
        hipMalloc((void**)&d_kernels, sizeof(float) * kernelSize * kernelCount);
        hipMalloc((void**)&d_paddedSignal, sizeof(hipfftComplex) * fftSize);
        hipMalloc((void**)&d_paddedKernels, sizeof(hipfftComplex) * fftSize * group_num);
        hipMalloc((void**)&d_results, sizeof(float) * fftSize * kernelCount);
        hipMalloc((void**)&d_signal_fft, cfftSize * sizeof(hipfftComplex));
        hipMalloc((void**)&d_kernels_fft, cfftSize * group_num * sizeof(hipfftComplex));
        hipMalloc((void**)&d_w_results, cfftSize * group_num * sizeof(hipfftComplex));

        // timer.restart();
        hipMemset(d_signal, 0, dataSize * sizeof(float));
        hipMemset(d_kernels, 0, kernelSize * kernelCount * sizeof(float));
        hipMemset(d_paddedSignal, 0, fftSize * sizeof(hipfftComplex));
        hipMemset(d_paddedKernels, 0, fftSize * group_num * sizeof(hipfftComplex));
        hipMemset(d_results, 0, fftSize * kernelCount * sizeof(float));
        // timerMap["memset"].push_back(timer.elapsed());

        // timer.restart();
        hipMemcpy(d_signal, Signal, dataSize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_kernels, Kernels, kernelSize * kernelCount * sizeof(float), hipMemcpyHostToDevice);
        //hipMemcpy(Kernels, d_kernels, kernelSize * kernelCount * sizeof(float), hipMemcpyDeviceToHost);
        //for(int i = 0;i<kernelSize;i++)
        //    printf("%f\n", Kernels[i]);
        //printf("\n");
        // timerMap["memcpy"].push_back(timer.elapsed());

        // padded signal
        dim3 threadBlock(16, 16, 4);
        dim3 dataBlockGrid(iDivUp(fftCols, threadBlock.x), iDivUp(fftRows, threadBlock.y), 1);
        dim3 kernelBlockGrid(iDivUp(fftCols, threadBlock.x), iDivUp(fftRows, threadBlock.y), iDivUp(group_num, threadBlock.z));
        //dim3 kernelBlockGrid(iDivUp(kCols, threadBlock.x), iDivUp(kRows, threadBlock.y), iDivUp(group_num, threadBlock.z));
        padDataWithZerosComplex<<<dataBlockGrid, threadBlock>>>(d_paddedSignal, d_signal, fftCols, fftRows, nCols, nRows, 1); 
        padKernelWithZerosComplex<<<kernelBlockGrid, threadBlock>>>(d_paddedKernels, d_kernels, fftCols, fftRows, kCols, kRows, group_num);

#if DEBUG
        CudaCheckError();
        hipfftComplex *h_paddedSignal = (hipfftComplex*)malloc(sizeof(hipfftComplex) * fftSize);
        hipfftComplex *h_paddedKernels = (hipfftComplex*)malloc(sizeof(hipfftComplex) * fftSize * group_num);

        hipMemcpy(h_paddedSignal, d_paddedSignal, sizeof(hipfftComplex) * fftSize, hipMemcpyDeviceToHost);
        hipMemcpy(h_paddedKernels, d_paddedKernels, sizeof(hipfftComplex) * fftSize * group_num, hipMemcpyDeviceToHost);

        for(int i = 0;i<fftRows;i++){
            for(int j = 0;j<fftCols;j++)
            	printf("%f+%fi ", h_paddedSignal[i * fftCols + j].x, h_paddedSignal[i * fftCols + j].y);
            printf("\n");
	}
        printf("\n");
        for(int i = 0;i<fftRows;i++){
            for(int j = 0;j<fftCols;j++)
            	printf("%f+%fi ", h_paddedKernels[i * fftCols + j].x, h_paddedKernels[i * fftCols + j].y);
            printf("\n");
	}
        printf("\n");
#endif
        hipfftPlan2d(&fwplan_signal, fftRows, fftCols, HIPFFT_C2C);
        hipfftPlanMany(&fwplan_kernels, 2, n_2d, inembed_2d, 1, idist_2d, onembed_2d, 1, odist_2d, HIPFFT_C2C, group_num);
        hipfftPlanMany(&bwplan, 2, n_2d, onembed_2d, 1, odist_2d, inembed_2d, 1, idist_2d, HIPFFT_C2C, group_num);

	// create 1d plan
        hipfftPlanMany(&plan1d, 1, n, inembed, 1, idist, onembed, 1, odist, HIPFFT_C2C, group_num * fftCols * 3 / 2);

#if DEBUG
        printf("finish plan construction.\n");
#endif
	// warm up
	{

            hipfftExecC2C(fwplan_signal, d_paddedSignal, d_signal_fft, HIPFFT_FORWARD);
            //hipfftExecC2C(fwplan_kernels, d_paddedKernels, d_kernels_fft, HIPFFT_FORWARD);
            //fast_fft(d_paddedSignal, 1, fftCols, fftRows, kCols, kRows, d_signal_fft, true);
            fast_fft(d_paddedKernels, group_num, fftCols, fftRows, kCols, kRows, d_kernels_fft, true);

            dim3 dotBlock(32, 32);
            dim3 dotGrid(fftCols / 32, fftCols / 32, 8);
			dotProduct<<<dotGrid, dotBlock>>>(d_signal_fft, fftCols, fftRows, 1.0 / fftSize, group_num/8, d_kernels_fft);

            // thrust::device_ptr<hipfftComplex> thr_signal_fft(d_signal_fft);
            // thrust::device_ptr<hipfftComplex> thr_kernel_fft(d_kernels_fft);
            // //thrust::transform(thr_signal_fft, thr_signal_fft + group_num * cfftSize, thr_kernel_fft, thr_kernel_fft, complexMultiplier(1.0 / fftSize));
            // for(int i = 0 ;i < group_num; i++){
            //     thrust::transform(thr_signal_fft, thr_signal_fft + cfftSize, thr_kernel_fft + i * cfftSize, thr_kernel_fft + i * cfftSize,
            //                       complexMultiplier(1.0 / fftSize));
            //     // timerMap["multiply"].push_back(timer.elapsed());
            // }

#if VERBOSE
            size_t freeMem, totalMem;
            hipMemGetInfo(&freeMem, &totalMem);
            std::cout << "Free: " << freeMem << ", Total: " << totalMem << std::endl;
#endif

            //hipMalloc((void**)&d_conj_w_results, cfftSize * group_num * sizeof(hipfftComplex));
            // timer.restart();
            hipfftExecC2C(bwplan, d_kernels_fft, d_w_results, HIPFFT_BACKWARD);
            //fast_fft(d_kernels_fft, group_num, fftCols, fftRows, d_w_results, false);
	}


        hipDeviceSynchronize();

	// timing record
	int fft_time = 0;
	int dot_product_time = 0;
	int ifft_time = 0;

	    //int duplicate = 8;
		//hipfftComplex* d_double_signal_fft;
        //hipMalloc((void**)&d_double_signal_fft, cfftSize * duplicate * sizeof(hipfftComplex));
        int num_repeats = NUM_ITERATIONS;
        for (int i = 0; i < num_repeats; ++i) 
	{
            auto start = std::chrono::steady_clock::now();
            hipfftExecC2C(fwplan_signal, d_paddedSignal, d_signal_fft, HIPFFT_FORWARD);
            //hipfftExecC2C(fwplan_kernels, d_paddedKernels, d_kernels_fft, HIPFFT_FORWARD);
            //fast_fft(d_paddedSignal, 1, fftCols, fftRows, d_signal_fft, true);
            fast_fft(d_paddedKernels, group_num, fftCols, fftRows, kCols, kRows, d_kernels_fft, true);
            auto end = std::chrono::steady_clock::now();
            fft_time += static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() );

            // timerMap["forward"].push_back(timer.elapsed());

            // timer.restart();

	    start = std::chrono::steady_clock::now();
            dim3 dotBlock(32, 32);
            dim3 dotGrid(fftCols / 32, fftCols / 32, 8);
			dotProduct<<<dotGrid, dotBlock>>>(d_signal_fft, fftCols, fftRows, 1.0 / fftSize, group_num/8, d_kernels_fft);
			//for(int c = 0; c < duplicate;c++)
            //    hipMemcpy(d_double_signal_fft + c * fftSize, d_signal_fft, sizeof(hipfftComplex) * fftSize, hipMemcpyDeviceToDevice);
            //thrust::device_ptr<hipfftComplex> thr_signal_fft(d_double_signal_fft);
            //thrust::device_ptr<hipfftComplex> thr_kernel_fft(d_kernels_fft);
            ////thrust::transform(thr_signal_fft, thr_signal_fft + group_num * cfftSize, thr_kernel_fft, thr_kernel_fft, complexMultiplier(1.0 / fftSize));
            //for(int i = 0 ;i < group_num / duplicate; i++){
            //    thrust::transform(thr_signal_fft, thr_signal_fft + duplicate * fftSize, thr_kernel_fft + i * duplicate * fftSize, thr_kernel_fft + i * duplicate * fftSize,
            //                      complexMultiplier(1.0 / fftSize));
            //    // timerMap["multiply"].push_back(timer.elapsed());
            //}
            hipDeviceSynchronize();
	    end = std::chrono::steady_clock::now();
            dot_product_time += static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() );

#if VERBOSE
            size_t freeMem, totalMem;
            hipMemGetInfo(&freeMem, &totalMem);
            std::cout << "Free: " << freeMem << ", Total: " << totalMem << std::endl;
#endif

            //hipMalloc((void**)&d_conj_w_results, cfftSize * group_num * sizeof(hipfftComplex));
            // timer.restart();
	    start = std::chrono::steady_clock::now();
            //fast_fft(d_kernels_fft, group_num, fftCols, fftRows, d_w_results, false);
            hipfftExecC2C(bwplan, d_kernels_fft, d_w_results, HIPFFT_BACKWARD);
            hipDeviceSynchronize();
	    end = std::chrono::steady_clock::now();
            ifft_time += static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() );

	}

        int avg_fft_time = fft_time / num_repeats;
        int avg_dot_product_time = dot_product_time / num_repeats;
        int avg_ifft_time = ifft_time / num_repeats;
	int avg_total_time = avg_fft_time + avg_dot_product_time + avg_ifft_time;
	printf("turbo_fft_gpu_v3(us)----fft:%d, dot_prod:%d, ifft:%d, total:%d\n", avg_fft_time, avg_dot_product_time, avg_ifft_time, avg_total_time);

        // split complex results
        SplitComplex<<<kernelBlockGrid, threadBlock>>>(d_results, d_w_results, fftCols, fftRows, group_num);
        //ConjComplex<<<kernelBlockGrid, threadBlock>>>(d_conj_w_results, d_w_results, fftCols, fftRows, group_num);
        //hipfftExecZ2Z(*bwplan, d_kernels_fft, d_kernels, HIPFFT_BACKWARD);
        // timerMap["backward"].push_back(timer.elapsed());

        // timer.restart();
        hipMemcpy(results, d_results, kernelCount * fftSize * sizeof(float), hipMemcpyDeviceToHost);
       
        // timerMap["copy_back"].push_back(timer.elapsed());
        // timerMap["total_time"].push_back(globalTimer.elapsed());

        // free GPU memory
        hipFree(d_signal);
        hipFree(d_kernels);
        hipFree(d_paddedSignal);
        hipFree(d_paddedKernels);
        hipFree(d_results);
        hipFree(d_signal_fft);
        hipFree(d_kernels_fft);
        hipFree(d_w_results);
        // free GPU memory for qdVectors
        hipFree(d_w_results);
        hipFree(d_qdVectors);

    }
    // direct fft for complex kernels
    void direct_fft(float * Kernels, int kernelCount, 
                             int nRows, int nCols, int kRows, int kCols, hipfftComplex * results)
    {

        hipfftHandle fwplan_kernels, bwplan; 

	int group_num = kernelCount / 2;
        int fftRows = nRows + kRows - 1;
        int fftCols = nCols + kCols - 1;
        int dataSize = nRows * nCols;
        int kernelSize = kRows * kCols;
        int fftSize = fftRows * fftCols;

	// parameters for cufftPlanMany1d
        int idist = fftCols * fftRows;
        int odist = fftCols * fftRows;
        int inembed[] = {fftCols, fftRows};
        int onembed[] = {fftCols, fftRows};
        int n[2] = {fftCols, fftRows};

#if DEBUG
        for(int i = 0;i<kRows;i++){
            for(int j = 0;j<kCols;j++)
                printf("%f ", Kernels[i * kCols + j]);
            printf("\n");
	}
        printf("\n");
#endif

        // device memory pointer
        float *d_kernels = NULL;  // device storage of kernels and final results
        hipfftComplex *d_paddedKernels = NULL;  // device storage of kernels and final results
        hipfftComplex *d_kernels_fft;
        hipfftComplex *d_results;
        
        hipMalloc((void**)&d_kernels, sizeof(float) * kernelSize * kernelCount);
        hipMalloc((void**)&d_paddedKernels, sizeof(hipfftComplex) * fftSize * group_num);
        hipMalloc((void**)&d_kernels_fft, sizeof(hipfftComplex) * fftSize * group_num);
        hipMalloc((void**)&d_results, fftSize * group_num * sizeof(hipfftComplex));

        // timer.restart();
        hipMemset(d_kernels, 0, kernelSize * kernelCount * sizeof(float));
        hipMemset(d_paddedKernels, 0, fftSize * group_num * sizeof(hipfftComplex));
        hipMemset(d_kernels_fft, 0, fftSize * group_num * sizeof(hipfftComplex));
        hipMemset(d_results, 0, fftSize * group_num * sizeof(hipfftComplex));
        // timerMap["memset"].push_back(timer.elapsed());

        hipMemcpy(d_kernels, Kernels, kernelSize * kernelCount * sizeof(float), hipMemcpyHostToDevice);
        // padded signal
        dim3 threadBlock(16, 16, 4);
        dim3 dataBlockGrid(iDivUp(fftCols, threadBlock.x), iDivUp(fftRows, threadBlock.y), 1);
        dim3 kernelBlockGrid(iDivUp(fftCols, threadBlock.x), iDivUp(fftRows, threadBlock.y), iDivUp(group_num, threadBlock.z));
        padKernelWithZerosComplex<<<kernelBlockGrid, threadBlock>>>(d_paddedKernels, d_kernels, fftCols, fftRows, kCols, kRows, group_num);

#if DEBUG
        CudaCheckError();
        hipfftComplex *h_paddedKernels = (hipfftComplex*)malloc(sizeof(hipfftComplex) * fftSize * group_num);

        hipMemcpy(h_paddedKernels, d_paddedKernels, sizeof(hipfftComplex) * fftSize * group_num, hipMemcpyDeviceToHost);

        for(int i = 0;i<fftRows;i++){
            for(int j = 0;j<fftCols;j++)
            	printf("%f+%fi ", h_paddedKernels[i * fftCols + j].x, h_paddedKernels[i * fftCols + j].y);
            printf("\n");
	}
        printf("\n");
#endif

	// create 2d plan
        hipfftPlanMany(&fwplan_kernels, 2, n, inembed, 1, idist, onembed, 1, odist, HIPFFT_C2C, group_num);
        hipfftExecC2C(fwplan_kernels, d_paddedKernels, d_kernels_fft, HIPFFT_BACKWARD);
        hipDeviceSynchronize();
        hipMemcpy(results, d_kernels_fft, group_num * fftSize * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
        //hipMemcpy(results, d_results, kernelCount * fftSize * sizeof(float), hipMemcpyDeviceToHost);

#if DEBUG
        for(int i = 0;i<kRows;i++){
            for(int j = 0;j<kCols;j++)
                printf("%f ", results[i * kCols + j].y);
            printf("\n");
	}
        printf("\n");
#endif

        // free GPU memory
        hipFree(d_kernels);
        hipFree(d_paddedKernels);
        hipFree(d_results);
        hipFree(d_kernels_fft);

    }

    // test fft with QD algorithm
    void qd_fft(float * Kernels, int kernelCount, 
                             int nRows, int nCols, int kRows, int kCols, hipfftComplex * results)
    {

        hipfftHandle fwplan_kernels, bwplan; 

	int group_num = kernelCount / 2;
        int fftRows = nRows + kRows - 1;
        int fftCols = nCols + kCols - 1;
        int dataSize = nRows * nCols;
        int kernelSize = kRows * kCols;
        int fftSize = fftRows * fftCols;
        int cfftSize = fftRows * fftCols;

	// parameters for cufftPlanMany1d
        int idist = fftCols;
        int odist = fftCols;
        int inembed[] = {0};
        int onembed[] = {0};
        int n[1] = {fftCols};

#if DEBUG
        for(int i = 0;i<kRows;i++){
            for(int j = 0;j<kCols;j++)
                printf("%f ", Kernels[i * kCols + j]);
            printf("\n");
	}
        printf("\n");
#endif

        // device memory pointer
        float *d_kernels = NULL;  // device storage of kernels and final results
        hipfftComplex *d_paddedKernels = NULL;  // device storage of kernels and final results
        hipfftComplex *d_w_results = NULL;
        hipfftComplex *d_results = NULL;
        
        hipMalloc((void**)&d_kernels, sizeof(float) * kernelSize * kernelCount);
        hipMalloc((void**)&d_paddedKernels, sizeof(hipfftComplex) * fftSize * group_num);
        hipMalloc((void**)&d_w_results, fftSize * 3 / 2 * group_num * sizeof(hipfftComplex));
        hipMalloc((void**)&d_results, fftSize * group_num * sizeof(hipfftComplex));

        // timer.restart();
        hipMemset(d_kernels, 0, kernelSize * kernelCount * sizeof(float));
        hipMemset(d_paddedKernels, 0, fftSize * group_num * sizeof(hipfftComplex));
        hipMemset(d_w_results, 0, fftSize * 3 / 2 * group_num * sizeof(hipfftComplex));
        // timerMap["memset"].push_back(timer.elapsed());

        hipMemcpy(d_kernels, Kernels, kernelSize * kernelCount * sizeof(float), hipMemcpyHostToDevice);
#if DEBUG
        printf("begin padding.\n");
#endif

        // padded signal
        dim3 threadBlock(16, 16, 4);
        dim3 dataBlockGrid(iDivUp(fftCols, threadBlock.x), iDivUp(fftRows, threadBlock.y), 1);
        dim3 kernelBlockGrid(iDivUp(fftCols, threadBlock.x), iDivUp(fftRows, threadBlock.y), iDivUp(group_num, threadBlock.z));
        padKernelWithZerosComplex<<<kernelBlockGrid, threadBlock>>>(d_paddedKernels, d_kernels, fftCols, fftRows, kCols, kRows, group_num);

	fast_fft(d_paddedKernels, group_num, fftCols, fftRows, kCols, kRows, d_results, false);
	hipMemcpy(results, d_results, sizeof(hipfftComplex) * fftSize * group_num, hipMemcpyDeviceToHost);
//#if DEBUG
//        printf("begin qd fft.\n");
//#endif
//
//	// cpu version of QD fft
//        hipfftComplex *h_paddedKernels = (hipfftComplex*)malloc(sizeof(hipfftComplex) * fftSize * group_num);  // padded kernels
//        hipMemcpy(h_paddedKernels, d_paddedKernels, sizeof(hipfftComplex) * fftSize * group_num, hipMemcpyDeviceToHost);
//
//#if DEBUG
//        for(int i = 0;i<fftRows;i++){
//            for(int j = 0;j<fftCols;j++)
//            	printf("%f+%fi ", h_paddedKernels[i * fftCols + j].x, h_paddedKernels[i * fftCols + j].y);
//            printf("\n");
//	}
//        printf("\n");
//#endif
//
//	// initilize p and s
//	int* p = new int[fftCols + fftCols / 2];
//	int* s = new int[fftCols + fftCols / 2];
//	for(int i = 0 ; i < fftCols; i ++){
//	    p[i] = 1;
//	    s[i] = i;
//	}
//	for(int i = fftCols; i < fftCols + fftCols / 2;i++){
//	    p[i] = (i - fftCols) * 2;
//	    s[i] = 1;
//	}
//
//#if DEBUG
//        printf("finish initilization of (p, s).\n");
//	for(int i = 0 ; i < fftCols + fftCols / 2;i++)
//	    printf("%d: p:%d, s:%d.\n", i, p[i], s[i]);
//#endif
//
//	// initilize indices of get data, 1.5 * N * 2N^2, indices of out data, 
//	int N = fftCols;
//	int* input_xs = new int[N * 3 / 2 * (N * N)];
//	int* input_ys = new int[N * 3 / 2 * (N * N)];
//	int* output_xs = new int[N * 3 / 2 * N];
//	int* output_ys = new int[N * 3 / 2 * N];
//
//        for(int i = 0; i < fftCols + fftCols/2;i++){
//	    for(int t = 0 ; t < fftCols;t++){
//		output_xs[i * fftCols + t] = (t * p[i]) % fftCols;
//		output_ys[i * fftCols + t] = (t * s[i]) % fftCols;
//	    }
//	}
//
//	for(int i = 0; i < fftCols + fftCols/2;i++)
//	    for(int t = 0; t < fftCols; t++){
//		int offset = i * fftSize + t * fftCols;
//	        find_input_index(p[i], s[i], t, fftCols, input_xs + offset, input_ys + offset);
//
////#if DEBUG
////		for(int j = 0; j < fftCols;j++)
////		    printf("%d, %d.\n", input_xs[offset + j], input_ys[offset + j]);
////		getchar();
////#endif
//	    }
//        
//#if DEBUG
//        printf("finish indices search.\n");
//#endif
//
//	hipfftComplex *h_qdVectors = (hipfftComplex*)malloc(sizeof(hipfftComplex) * fftSize * 3 / 2 * group_num);
//        memset(h_qdVectors, 0, sizeof(hipfftComplex) * fftSize * 3 / 2 * group_num);
//	// construct 1d vectors for 1d-fft
//	for(int k = 0; k < group_num;k++){
//	    int kOffset = k * fftSize * 3 / 2;
//            for(int i = 0; i < fftCols + fftCols/2;i++)
//	        for(int t = 0; t < fftCols; t++)
//	            for(int j = 0; j < fftCols;j++){
//	    	        int offset = i * fftSize + t * fftCols + j;
//	    	        int idx_x = input_xs[offset];
//	    	        int idx_y = input_ys[offset];
//	                h_qdVectors[kOffset + i * fftCols + t].x += h_paddedKernels[k * fftSize + idx_x * fftCols + idx_y].x;
//	                h_qdVectors[kOffset + i * fftCols + t].y += h_paddedKernels[k * fftSize + idx_x * fftCols + idx_y].y;
//	    	}
//	}
//
//#if DEBUG
//        printf("finish vector construction.\n");
//#endif
//
//        hipfftComplex *d_qdVectors;
//        hipMalloc((void**)&d_qdVectors, sizeof(hipfftComplex) * fftSize * 3 / 2 * group_num);
//	hipMemcpy(d_qdVectors, h_qdVectors, sizeof(hipfftComplex) * fftSize * 3 / 2 * group_num, hipMemcpyHostToDevice);
//	// create 1d plan
//        hipfftPlanMany(&fwplan_kernels, 1, n, inembed, 1, idist, onembed, 1, odist, HIPFFT_C2C, group_num * fftCols * 3 / 2);
//
//#if DEBUG
//        printf("finish plan construction.\n");
//#endif
//
//        hipfftExecC2C(fwplan_kernels, d_qdVectors, d_w_results, HIPFFT_FORWARD);
//
//        hipDeviceSynchronize();
//
//#if DEBUG
//        printf("finish fft.\n");
//#endif
//
//	hipfftComplex* h_w_results = (hipfftComplex*)malloc(group_num * fftSize * 3 / 2 * sizeof(hipfftComplex));
//        hipMemcpy(h_w_results, d_w_results, group_num * fftSize * 3 / 2 * sizeof(hipfftComplex), hipMemcpyDeviceToHost);
//
//        // re-arrange results
//	for(int k = 0; k < group_num;k++){
//	    int kOffset = k * fftSize * 3 / 2;
//            for(int i = 0; i < fftCols + fftCols/2;i++)
//	        for(int t = 0; t < fftCols; t++){
//		    int idx_x = output_xs[i * fftCols + t];
//		    int idx_y = output_ys[i * fftCols + t];
//		    results[k * fftSize + idx_x * fftCols + idx_y].x = h_w_results[kOffset + i * fftCols + t].x;
//		    results[k * fftSize + idx_x * fftCols + idx_y].y = h_w_results[kOffset + i * fftCols + t].y;
//		}
//		
//	}
//
//        // free GPU memory
//        hipFree(d_kernels);
//        hipFree(d_paddedKernels);
//        hipFree(d_w_results);
//        hipFree(d_qdVectors);
#if DEBUG
        CudaCheckError();
        for(int i = 0;i<fftRows;i++){
            for(int j = 0;j<fftCols;j++)
            	printf("%f+%fi ", results[i * fftCols + j].x, results[i * fftCols + j].y);
            printf("\n");
	}
        printf("\n");
#endif

    }
//}
